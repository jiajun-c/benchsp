#include "cusp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_runtime.h>

void cusparse_spmv(int *A_csrOffsets, int *A_columns, float *A_values, float *x, float* y, int row, int col, int nnz) {
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    hipMalloc((void**)&dA_csrOffsets, (row+1)*sizeof(int));
    hipMalloc((void**)&dA_columns, nnz*sizeof(int));
    hipMalloc((void**)&dA_values, nnz*sizeof(float));
    hipMalloc((void**)&dX, col*sizeof(float));
    hipMalloc((void**)&dY, row*sizeof(float));

    hipMemcpy(dA_csrOffsets, A_csrOffsets, (row + 1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_columns, A_columns, nnz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_values, A_values, nnz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX, x, col*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dY, y, row*sizeof(float), hipMemcpyHostToDevice);
    
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    hipsparseCreate(&handle);
    hipsparseCreateCsr(&matA, row, col, nnz,
        dA_csrOffsets, dA_columns, dA_values,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&vecX, col, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, row, dY, HIP_R_32F);
    float alpha = 1.0f, beta = 0.0f;
    size_t bufSize;
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          &alpha, matA, vecX, &beta, vecY,
                          HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufSize);
    float *dBuffer;
    hipMalloc(&dBuffer, bufSize);
    hipsparseSpMV_preprocess(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);
    hipMemcpy(y, dY, row * sizeof(float),hipMemcpyDeviceToHost);
}