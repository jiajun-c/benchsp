#include "cusp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_runtime.h>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}
int cusparse_spmv(int *hA_csrOffsets, int64_t *hA_columns, float *hA_values, float *hX, float* hY, int64_t A_num_rows, int64_t A_num_cols, int64_t A_nnz) {
    // float     alpha           = 1.0f;
    // float     beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    // const int A_num_rows      = 4;
    // const int A_num_cols      = 4;
    // const int A_nnz           = 9;
    // int       hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
    // int       hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    // float     hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
    //                               6.0f, 7.0f, 8.0f, 9.0f };
    // float     hX[]            = { 1.0f, 1.0f, 1.0f, 1.0f };
    // float     hY[]            = { 0.0f, 0.0f, 0.0f, 0.0f };
    // float     hY_result[]     = { 19.0f, 8.0f, 51.0f, 52.0f };
    // hY[0] = 3.0;
    // printf("A_num_rows %d %d %d\n", A_num_rows, A_num_cols, A_nnz);
    // for (int i = 0; i < 5; i++) {
    //     printf("A_csrOffsets %d %d\n", i, hA_csrOffsets[i]);

    // }
    // for (int i = 0; i < A_nnz; i++) {
    //     printf("hA_columns %d %d\n", i, hA_columns[i]);

    // }
    //     for (int i = 0; i < A_nnz; i++) {
    //     printf("hA_values %d %f\n", i, hA_values[i]);

    // }
    //     for (int i = 0; i < 4; i++) {
    //     printf("hX %d %f\n", i, hX[i]);

    // }
    float     alpha           = 1.0f;
    float     beta            = 0.0f;
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         A_num_cols * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         A_num_rows * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, hX, A_num_cols * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, hY, A_num_rows * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_32F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    // CHECK_CUSPARSE( hipsparseSpMV_preprocess(
    //                              handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                              &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
    //                              HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hY, dY, A_num_rows * sizeof(float),hipMemcpyDeviceToHost) )
    // hY[0] = 3.0;
    // for (int i = 0; i < A_num_rows; i++)
    // printf("%f\n", hY[i]);
}