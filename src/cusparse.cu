#include "cusp.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_runtime.h>
#include <chrono>
#include <hip/hip_fp16.h>
#include <iostream>
#include "matrixFormat.hpp"
#include "../utils/utils.hpp"
#include "hip/hip_fp16.h"
#include <iostream>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}
int cusparse_spmv(int *hA_csrOffsets, int64_t *hA_columns, float *hA_values, float *hX, float* hY, int64_t A_num_rows, int64_t A_num_cols, int64_t A_nnz) {
    float     alpha           = 1.0f;
    float     beta            = 0.0f;
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         A_num_cols * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         A_num_rows * sizeof(float)) )
    
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, hX, A_num_cols * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, hY, A_num_rows * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_32F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSpMV_preprocess(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hY, dY, A_num_rows * sizeof(float),hipMemcpyDeviceToHost) )
    return 0;
}

int cusparse_spmv_fp64(int *hA_csrOffsets, int64_t *hA_columns, double *hA_values, double *hX, double* hY, int64_t A_num_rows, int64_t A_num_cols, int64_t A_nnz) {
    double     alpha           = 1.0f;
    double     beta            = 0.0f;
    int   *dA_csrOffsets, *dA_columns;
    double *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(double))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         A_num_cols * sizeof(double)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         A_num_rows * sizeof(double)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(double),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, hX, A_num_cols * sizeof(double),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, hY, A_num_rows * sizeof(double),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_64F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_64F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSpMV_preprocess(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // execute SpMV
    auto start = std::chrono::high_resolution_clock::now(); 

    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )
    auto end = std::chrono::high_resolution_clock::now(); 
    auto elapsed = end - start;
    std::cout << "耗时: " << std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count() << " us\n";
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hY, dY, A_num_rows * sizeof(double),hipMemcpyDeviceToHost) )
    return 0;
}


int cusparse_spmv_fp16(int *hA_csrOffsets, int64_t *hA_columns, half *hA_values, half *hX, half* hY, int64_t A_num_rows, int64_t A_num_cols, int64_t A_nnz) {
    half     alpha           = 1.0f;
    half     beta            = 0.0f;
    int   *dA_csrOffsets, *dA_columns;
    half *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(half))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         A_num_cols * sizeof(half)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         A_num_rows * sizeof(half)) )
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(half),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, hX, A_num_cols * sizeof(half),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, hY, A_num_rows * sizeof(half),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_64F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_64F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSpMV_preprocess(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // execute SpMV
    auto start = std::chrono::high_resolution_clock::now(); 

    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )
    auto end = std::chrono::high_resolution_clock::now(); 
    auto elapsed = end - start;
    std::cout << "耗时: " << std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count() << " us\n";
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hY, dY, A_num_rows * sizeof(half),hipMemcpyDeviceToHost) )
    return 0;
}

int cusparse_spmm_fp16(int *hA_csrOffsets, int32_t *hA_columns, float *hA_values32,float *hB32, float* hC32, int M, int N, int K, int A_nnz) {
    int   *dA_csrOffsets, *dA_columns;
    half* hA_values, *hB, *hC;
    half *dA_values, *dB, *dC;
    int A_num_rows = M;
    int A_num_cols = K;
    // int B_size = N * K;
    // int C_size = M * N;
    int   B_num_rows      = A_num_cols;
    int   B_num_cols      = N;
    int   ldb             = B_num_rows;
    int   ldc             = A_num_rows;
    int   B_size          = ldb * B_num_cols;
    int   C_size          = ldc * B_num_cols;
    half alpha           = 1.0f;
    half beta            = 0.0f;
    hA_values = (half*) malloc(A_nnz * sizeof(half));
    for (int i = 0; i < A_nnz; i++) {
        hA_values[i] = (half)hA_values32[i];
    }

    hB = (half*) malloc(B_size * sizeof(half));
    for (int i = 0; i < B_size; i++) {
        hB[i] = (half)hB32[i];
    }

    hC = (half*) malloc(C_size * sizeof(half));
    for (int i = 0; i < C_size; i++) {
        hC[i] = (half)hC32[i];
    }
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(half))  )
    CHECK_CUDA( hipMalloc((void**) &dB,         B_size * sizeof(half)) )
    CHECK_CUDA( hipMalloc((void**) &dC,         C_size * sizeof(half)) )
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(half),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(half),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size * sizeof(half),
                           hipMemcpyHostToDevice) )
    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                     dA_csrOffsets, dA_columns, dA_values,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) )
    // CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, N, K, K, dB, HIP_R_16F,
    //                                   HIPSPARSE_ORDER_ROW) )
    // CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, M, N, M, dC, HIP_R_16F,
    //                                   HIPSPARSE_ORDER_ROW) )


    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
        HIP_R_16F, HIPSPARSE_ORDER_COL) )
// Create dense matrix C
        CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
            HIP_R_16F, HIPSPARSE_ORDER_COL) )
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                        handle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    CHECK_CUSPARSE( hipsparseSpMM_preprocess(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    auto start = std::chrono::high_resolution_clock::now(); 
    CHECK_CUSPARSE( hipsparseSpMM(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
        auto end = std::chrono::high_resolution_clock::now(); 
        auto elapsed = end - start;
        std::cout << "cusparse 耗时: " << std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count() << " us\n";
       
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    CHECK_CUDA( hipMemcpy(hC, dC, C_size * sizeof(half), hipMemcpyDeviceToHost) )
    for (int i = 0; i < C_size; i++) {
        hC32[i] = (float)hC[i];
    }
    return 0;                                 
}