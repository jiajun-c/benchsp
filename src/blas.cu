#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
using namespace std;

int gemvfp64(double *A, double *B, double *C, int M, int N, int K, int repeat) {
    hipblasStatus_t status;
    double alpha = 1.0;
    double beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, M * N * sizeof(double));
    hipMalloc((void **)&d_B, N * sizeof(double));
    hipMalloc((void **)&d_C, M * sizeof(double));
    hipMemcpy(d_A, A, M * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, M * sizeof(double), hipMemcpyHostToDevice);
    for (int i = 0; i < repeat; i++) {
        auto start = chrono::high_resolution_clock::now();
        status = hipblasDgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_B, 1, &beta, d_C, 1);
        hipDeviceSynchronize();
        auto end = chrono::high_resolution_clock::now();
        auto elapsed = end - start;
        std::cout << "dense blas 耗时: " << std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count() << " us\n";
    }
    hipMemcpy(C, d_C, M * sizeof(double), hipMemcpyDeviceToHost);
    return status;
}